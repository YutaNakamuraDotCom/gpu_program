
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromeGPU(){
    printf("Hello World from GPU! %d\n",threadIdx.x);
}

int main(int argc, char **argv){
    printf("Hello World from CPU!\n");
    helloFromeGPU<<<1,100>>>();
    hipDeviceReset();
    //cudaDeviceSynchronize();
    return 0;
}
